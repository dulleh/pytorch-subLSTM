#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}