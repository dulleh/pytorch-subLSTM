#include "hip/hip_runtime.h"
/**
  * This is all from https://pytorch.org/tutorials/advanced/cpp_extension.html
  */
#include "sublstm.cuh"

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}

template <typename scalar_t>
__global__ void forward_cuda_kernel(
    //TODO: I changed this 3->4 because we needed a forget gate?
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> gates,
    const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> old_cell,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_h,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_cell,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input_gate,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output_gate,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> forget_gate,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> candidate_cell) {
  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < gates.size(2)){
  //TODO: We need a forget gate, but also need to check if these are ordered correctly
    input_gate[n][c] = sigmoid(gates[n][0][c]);
    output_gate[n][c] = sigmoid(gates[n][1][c]);
    candidate_cell[n][c] = sigmoid(gates[n][2][c]);
    forget_gate[n][c] = sigmoid(gates[n][3][c]);
    new_cell[n][c] =
        old_cell[n][c] * forget_gate + candidate_cell[n][c] - input_gate[n][c];
    new_h[n][c] = sigmoid(new_cell[n][c]) - output_gate[n][c];
  }
}

std::vector<torch::Tensor> forward_cuda(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor old_h,
    torch::Tensor old_cell) {
  auto X = torch::cat({old_h, input}, /*dim=*/1);
  auto gates = torch::addmm(bias, X, weights.transpose(0, 1));

  const auto batch_size = old_cell.size(0);
  const auto state_size = old_cell.size(1);

  auto new_h = torch::zeros_like(old_cell);
  auto new_cell = torch::zeros_like(old_cell);
  auto input_gate = torch::zeros_like(old_cell);
  auto output_gate = torch::zeros_like(old_cell);
  auto forget_gate = torch::zeros_like(old_cell);
  auto candidate_cell = torch::zeros_like(old_cell);

  /**
    * As for the kernel launch itself, we are here specifying that each CUDA block will have 1024 threads, and that the
    * entire GPU grid is split into as many blocks of 1 x 1024 threads as are required to fill our matrices with one
    * thread per component. For example, if our state size was 2048 and our batch size 4, we’d launch a total of
    * 4 x 2 = 8 blocks with each 1024 threads.=
    * Source: https://pytorch.org/tutorials/advanced/cpp_extension.html#writing-a-mixed-c-cuda-extension
    **/
  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(gates.type(), "sublstm_forward_cuda", ([&] {
    forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
        gates.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        input_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        output_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        candidate_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));

  return {new_h, new_cell, input_gate, output_gate, forget_gate, candidate_cell, X, gates};
}