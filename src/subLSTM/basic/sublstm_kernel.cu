#include "hip/hip_runtime.h"
/**
  * This is all from https://pytorch.org/tutorials/advanced/cpp_extension.html
  * Taking cues from  	https://github.com/pytorch/extension-cpp/blob/master/cuda/lltm_cuda_kernel.cu
  */

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <stdio.h>

namespace {

	template <typename scalar_t>
	__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
	  return 1.0 / (1.0 + exp(-z));
	}

	template <typename scalar_t>
	__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
	  const auto s = sigmoid(z);
	  return (1.0 - s) * s;
	}

	template <typename scalar_t>
	__global__ void forward_cuda_kernel(
		const int X_size,
		const int batch_size,
		const int state_size,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> X,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights,
		const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> bias,
		torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> gates,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> old_cell,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_h,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_cell,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input_gate,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output_gate,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> forget_gate,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> candidate_cell)
		{
	  //batch index
	  const int n = blockIdx.y;
	  // column index ie output state index
	  const int c = blockIdx.x * blockDim.x + threadIdx.x;
	  if (c < state_size) {
	  	for (int k = 0; k < X_size; k++) {
		  	gates[n][0][c] += X[n][k] * weights[c][k];
	      gates[n][1][c] += X[n][k] * weights[state_size + c][k];
		  	gates[n][2][c] += X[n][k] * weights[2*state_size + c][k];
		  	gates[n][3][c] += X[n][k] * weights[3*state_size + c][k];
			}
			gates[n][0][c] += bias[c];
			gates[n][1][c] += bias[state_size + c];
			gates[n][2][c] += bias[2*state_size + c];
			gates[n][3][c] += bias[3*state_size + c];

			input_gate[n][c] = sigmoid(gates[n][0][c]);
			output_gate[n][c] = sigmoid(gates[n][1][c]);
			candidate_cell[n][c] = sigmoid(gates[n][2][c]);
			forget_gate[n][c] = sigmoid(gates[n][3][c]);
			new_cell[n][c] =
				(old_cell[n][c] * forget_gate[n][c]) + (candidate_cell[n][c] - input_gate[n][c]);
			new_h[n][c] = sigmoid(new_cell[n][c]) - output_gate[n][c];
	  }
	}

	template <typename scalar_t>
	__global__ void backward_cuda_kernel(
		const int batch_size,
		const int state_size,
		const int input_size,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> grad_h,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_cell,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> grad_cell,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> forget_gate,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> X,
		const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> gate_weights,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> old_cell,
		const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> weights,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_old_cell,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_gates,
		torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> d_X_intermediates,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_old_h,
		torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_input)
	{
	  // batch index
	  const int n = blockIdx.y;
	  // column index ie output state index
	  const int c = blockIdx.x * blockDim.x + threadIdx.x;
	  if (c < state_size) {
	  	const auto d_new_cell = (grad_h[n][c] * d_sigmoid(new_cell[n][c])) + grad_cell[n][c];
	  	d_old_cell[n][c] = d_new_cell * forget_gate[n][c];
	  	d_gates[n][c] = -d_new_cell * d_sigmoid(gate_weights[n][0][c]); // d_input_gate pre-activation
	  	d_gates[n][state_size + c] = -grad_h[n][c] * d_sigmoid(gate_weights[n][1][c]); // d_output_gate  pre-activation
	  	d_gates[n][2*state_size + c] = d_new_cell * d_sigmoid(gate_weights[n][2][c]); // d_candidate_cell pre-activation
	  	d_gates[n][3*state_size + c] = (d_new_cell * old_cell[n][c]) * d_sigmoid(gate_weights[n][3][c]); // d_forget_gate pre-activation

			__syncthreads();
			for (int k = 0; k < state_size + input_size; k++) {
				d_X_intermediates[c] = d_gates[n][c] * weights[c][k]
										+ d_gates[n][state_size + c] * weights[state_size + c][k]
										+ d_gates[n][2*state_size + c] * weights[2*state_size + c][k]
										+ d_gates[n][3*state_size + c] * weights[3*state_size + c][k];
				// synchronize, then one thread sums up the intermediate sums
				__syncthreads();
				if (c == 0) {
					if (k < state_size) {
						for (int i = 0; i < state_size; i++) {
								d_old_h[n][k] += d_X_intermediates[i];
						}
					} else {
						for (int i = 0; i < state_size; i++) {
								d_input[n][k] += d_X_intermediates[i];
						}
					}
				}
			}
			//atomicAdd_block();
		}
	}

}

std::vector<torch::Tensor> forward_cuda(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor old_h,
    torch::Tensor old_cell)
{
  auto X = torch::cat({old_h, input}, /*dim=*/1);

  //std::cout << "X: " << X << std::endl;
  //std::cout << "weights: " << weights << std::endl;
 // std::cout << "bias: " << bias << std::endl;

  const auto X_size = X.size(1);
  const auto batch_size = old_cell.size(0);
  const auto state_size = old_cell.size(1);

//  std::cout << "X_size: " << X_size << std::endl;
//  std::cout << "batch_size: " << batch_size << std::endl;
//  std::cout << "state_size: " << state_size << std::endl;

  auto gates = torch::zeros({batch_size, 4, state_size}, weights.options());
  auto new_h = torch::zeros_like(old_cell);
  auto new_cell = torch::zeros_like(old_cell);
  auto input_gate = torch::zeros_like(old_cell);
  auto output_gate = torch::zeros_like(old_cell);
  auto forget_gate = torch::zeros_like(old_cell);
  auto candidate_cell = torch::zeros_like(old_cell);

  /**
    * As for the kernel launch itself, we are here specifying that each CUDA block will have 1024 threads, and that the
    * entire GPU grid is split into as many blocks of 1 x 1024 threads as are required to fill our matrices with one
    * thread per component. For example, if our state size was 2048 and our batch size 4, we’d launch a total of
    * 4 x 2 = 8 blocks with each 1024 threads.=
    * Source: https://pytorch.org/tutorials/advanced/cpp_extension.html#writing-a-mixed-c-cuda-extension
    **/
  const int threads = 512;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "forward_cuda", ([&] {
    forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
		X_size,
		batch_size,
		state_size,
		X.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
		bias.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
	    gates.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
		old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        input_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        output_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        forget_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        candidate_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));

  return {new_h, new_cell, input_gate, output_gate, forget_gate, candidate_cell, X, gates};
}


std::vector<torch::Tensor> backward_cuda(
    torch::Tensor grad_h,
    torch::Tensor grad_cell,
    torch::Tensor new_cell,
    torch::Tensor input_gate, // these are the outputs of these gates
    torch::Tensor output_gate,
    torch::Tensor forget_gate,
    torch::Tensor candidate_cell,
    torch::Tensor X,
    torch::Tensor gate_weights, // gate outputs, pre-activation
    torch::Tensor weights, // actual weights in the gates
    torch::Tensor old_cell)
{
  const auto batch_size = grad_h.size(0);
  const auto state_size = grad_h.size(1);
	const auto input_size = X.size(1) - state_size;

	// auto d_new_cell  -- Don't need this as it is not returned, and used only within the kernel
	auto d_old_cell = torch::zeros_like(old_cell);
	auto d_gates = torch::zeros({batch_size, 4*state_size}, weights.options());

	auto d_X_intermediates = torch::zeros({state_size}, weights.options());
	auto d_old_h = torch::zeros({batch_size, state_size}, weights.options());
	auto d_input = torch::zeros({batch_size, input_size}, weights.options());

	const int threads = 512;
  const dim3 blocks((state_size + input_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(grad_h.scalar_type(), "backward_cuda", ([&] {
    backward_cuda_kernel<scalar_t><<<blocks, threads>>>(
      batch_size,
      state_size,
			input_size,
			grad_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
      new_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			grad_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			forget_gate.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			X.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			gate_weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
			old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			weights.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			d_old_cell.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			d_gates.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			d_X_intermediates.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
			d_old_h.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
			d_input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));

	//std::cout << "cu: d_gates[0][12]" << d_gates[1][1] << std::endl;
	//std::cout << "cu: d_old_cell[1][300]" << d_old_cell[1][300] << std::endl;


	//!!!!!!!! NEED TO SET THIS SIZE
	//auto d_weights = torch::zeros({4*state_size, X.size(1)}, weights.options());

	torch::Tensor d_weights = d_gates.t().mm(X);

	// sum across rows i.e. sum of columns,
	// keepdim=true means we're getting a result that has 1 row, columns same as before
	torch::Tensor d_bias = d_gates.sum(0, true); // not entirely sure why we're summing but I can see that the resulting shape is correct

	//torch::Tensor d_X = d_gates.mm(weights);
	//torch::Tensor d_old_h = d_X.slice(1, 0, state_size); // first state_size columns
	//torch::Tensor d_input = d_X.slice(1, state_size); // from column [state_size + 1] to the end

	return {d_old_h, d_input, d_weights, d_bias, d_old_cell, d_gates};
}
